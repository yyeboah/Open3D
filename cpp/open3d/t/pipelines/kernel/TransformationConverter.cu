#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "open3d/t/pipelines/kernel/TransformationConverterImpl.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

__global__ void PoseToTransformationKernel(float *transformation_ptr,
                                           const float *X_ptr) {
    PoseToTransformationImpl(transformation_ptr, X_ptr);
}

void PoseToTransformationCUDA(float *transformation_ptr, const float *X_ptr) {
    PoseToTransformationKernel<<<1, 1>>>(transformation_ptr, X_ptr);
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
